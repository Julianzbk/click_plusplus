#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <cstdlib>
#include <cstdio>

constexpr size_t threads_per_block = 256;

template <typename dtype = float>
__global__
void dot_vector(const dtype* A, const dtype* B, dtype* partial, size_t N)
{
    extern __shared__ dtype cache[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    cache[tid] = (i < N) ? A[i] * B[i] : dtype(); // operation step
    __syncthreads();
    
    //printf("hi from idx");
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            cache[tid] += cache[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {   
        partial[blockIdx.x] = cache[0];
    }
}


extern "C" float dot_vector_float(const float* h_A, const float* h_B, size_t N)
{
    /*
        Copies host vector to cudaMalloc'd vector, and performs dot product,
        accumulating partial sums and returning their sum.
    */
    const size_t blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
    const size_t size = N * sizeof(float);

    float *A, *B, *partial;
    hipMalloc(&A, size);
    hipMalloc(&B, size);
    hipMalloc(&partial, blocks_per_grid * sizeof(float));
    hipMemcpy(A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(B, h_B, size, hipMemcpyHostToDevice);

    dot_vector<float> <<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>
        (A, B, partial, N);
    
    float* h_partial = new float[blocks_per_grid];
    hipMemcpy(h_partial, partial, blocks_per_grid * sizeof(float), hipMemcpyDeviceToHost);
    float prod = 0;
    for (size_t i = 0; i < blocks_per_grid; ++i)
    {
        prod += h_partial[i];
    }

    hipFree(A);
    hipFree(B);
    hipFree(partial);
    delete[] h_partial;

    return prod;
}

extern "C" float device_dot_vector_float(const float* d_A, const float* d_B, size_t N)
{
    /*
        Performs dot product directly on device memory.
    */
    const size_t blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    float* partial;
    hipMalloc(&partial, blocks_per_grid * sizeof(float));

    dot_vector<float> <<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>
        (d_A, d_B, partial, N);
    
    float* h_partial = new float[blocks_per_grid];
    hipMemcpy(h_partial, partial, blocks_per_grid * sizeof(float), hipMemcpyDeviceToHost);

    float prod = 0;
    for (size_t i = 0; i < blocks_per_grid; ++i)
    {
        prod += h_partial[i];
    }

    hipFree(partial);
    delete[] h_partial;
    return prod;
}


template <class dtype>
__global__
void vector_add_vector(const dtype* V, const dtype* U, dtype* dest, size_t N)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    dest[i] = (i < N) ? V[i] + U[i] : dtype(); // operation step
}

extern "C" float* vector_add_vector_float(const float* V, const float* U, size_t N)
{// Transfers ownership of a live pointer!
    const size_t blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    float* dest;
    hipMalloc(&dest, N * sizeof(float));

    vector_add_vector<float> <<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>
        (V, U, dest, N);
    
    return dest;
}

template <class dtype>
__global__
void vector_addassign_vector(dtype* V, const dtype* U, size_t N)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    V[i] += (i < N) ? U[i] : dtype();
}

extern "C" void vector_addassign_vector_float(float* V, const float* U, size_t N)
{
    const size_t blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    vector_addassign_vector<float> <<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>
        (V, U, N);
}

template <class dtype>
__global__
void vector_sub_scalar(const dtype* V, dtype A, dtype* dest, size_t N)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    dest[i] += (i < N) ? V[i] - A : dtype();
}

extern "C" float* vector_sub_scalar_float(const float* V, float A, size_t N)
{
    const size_t blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    float* dest;
    hipMalloc(&dest, N * sizeof(float));

    vector_sub_scalar<float> <<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>
        (V, A, dest, N);
    return dest;
}

template <class dtype>
__global__
void vector_sub_vector(const dtype* V, const dtype* U, dtype* dest, size_t N)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    dest[i] = (i < N) ? V[i] - U[i] : dtype(); // operation step
}

extern "C" float* vector_sub_vector_float(const float* V, const float* U, size_t N)
{// Transfers ownership of a live pointer!
    const size_t blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    float* dest;
    hipMalloc(&dest, N * sizeof(float));

    vector_sub_vector<float> <<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>
        (V, U, dest, N);
    
    return dest;
}

template <class dtype>
__global__
void vector_subassign_vector(dtype* V, const dtype* U, size_t N)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    V[i] -= (i < N) ? U[i] : dtype();
}

extern "C" void vector_subassign_vector_float(float* V, const float* U, size_t N)
{
    const size_t blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    vector_subassign_vector<float> <<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>
        (V, U, N);
}

template <class dtype>
__global__
void vector_mul_scalar(const dtype* V, dtype A, dtype* dest, size_t N)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    dest[i] += (i < N) ? V[i] * A : dtype();
}

extern "C" float* vector_mul_scalar_float(const float* V, float A, size_t N)
{// Transfers ownership of a live pointer!
    const size_t blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    float* dest;
    hipMalloc(&dest, N * sizeof(float));

    vector_mul_scalar<float> <<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>
        (V, A, dest, N);
    
    return dest;
}

template <class dtype>
__global__
void vector_div_scalar(const dtype* V, dtype A, dtype* dest, size_t N)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    dest[i] += (i < N) ? V[i] / A : dtype();
}

extern "C" float* vector_div_scalar_float(const float* V, float A, size_t N)
{// Transfers ownership of a live pointer!
    const size_t blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    float* dest;
    hipMalloc(&dest, N * sizeof(float));

    vector_div_scalar<float> <<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>
        (V, A, dest, N);
    
    return dest;
}