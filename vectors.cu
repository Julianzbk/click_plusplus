#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <cstdlib>
#include <cstdio>

template <typename dtype = float>
__global__
void dot_product_vector(const dtype* A, const dtype* B, dtype* partial, size_t N)
{
    extern __shared__ dtype cache[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    cache[tid] = (i < N) ? A[i] * B[i] : dtype(); // operation step
    __syncthreads();
    
    //printf("hi from idx");
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            cache[tid] += cache[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {   
        partial[blockIdx.x] = cache[0];
    }
}

extern "C" float dot_product_vector_float(const float* h_A, const float* h_B, size_t N)
{
    /*
        Copies host vector to cudaMalloc'd vector, and performs dot product,
        accumulating partial sums and returning their sum.
    */
    constexpr size_t threads_per_block = 256;
    const size_t blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
    const size_t size = N * sizeof(float);

    float *A, *B, *partial;
    hipMalloc(&A, size);
    hipMalloc(&B, size);
    hipMalloc(&partial, blocks_per_grid * sizeof(float));
    hipMemcpy(A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(B, h_B, size, hipMemcpyHostToDevice);

    dot_product_vector<float> <<<blocks_per_grid, threads_per_block, threads_per_block * sizeof(float)>>>
        (A, B, partial, N);
    
    float* h_partial = new float[blocks_per_grid];
    hipMemcpy(h_partial, partial, blocks_per_grid * sizeof(float), hipMemcpyDeviceToHost);
    float prod = 0;
    for (size_t i = 0; i < blocks_per_grid; ++i)
    {
        prod += h_partial[i];
    }

    hipFree(A);
    hipFree(B);
    hipFree(partial);
    delete[] h_partial;

    return prod;
}